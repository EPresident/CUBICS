#include "hip/hip_runtime.h"
#include <algorithm>

#include <searchers/IntBacktrackStack.h>
#include <utils/KernelUtils.h>

void IntBacktrackStack::initialize(IntDomainsRepresentations* representations, Statistics* stats)
{
    this->representations = representations;

    backupsStacks.initialize(representations->bitvectors.size);
    backupsStacks.resize(representations->bitvectors.size);

    levelsStacks.initialize(representations->bitvectors.size);
    levelsStacks.resize(representations->bitvectors.size);

    for (int vi = 0; vi < backupsStacks.size; vi += 1)
    {
        backupsStacks[vi].initialize(VECTOR_INITIAL_CAPACITY);

        levelsStacks[vi].initialize();
    }

    this->stats = stats;
}

void IntBacktrackStack::deinitialize()
{
    for (int vi = 0; vi < backupsStacks.size; vi += 1)
    {
        backupsStacks[vi].deinitialize();
        levelsStacks[vi].deinitialize();
    }
    backupsStacks.deinitialize();
    levelsStacks.deinitialize();
}

cudaDevice void IntBacktrackStack::saveState(int backtrackLevel)
{
#ifdef GPU
    int vi = KernelUtils::getTaskIndex();
    if (vi >= 0 and vi < backupsStacks.size)
#else
    for (int vi = 0; vi < backupsStacks.size; vi += 1)
#endif
    {
        if (backtrackLevel == 0 || isDomainChanged(vi))
        {
            int min = representations->minimums[vi];
            int max = representations->maximums[vi];
            int offset = representations->offsets[vi];
            int version = representations->versions[vi];
            Vector<unsigned int>* bitvector = &representations->bitvectors[vi];
            backupsStacks[vi].push(min, max, offset, version, bitvector);

            levelsStacks[vi].push_back(backtrackLevel);

            stats->maxStackSize = std::max(stats->maxStackSize, levelsStacks[vi].size - 1);
        }
    }
}

cudaDevice void IntBacktrackStack::restoreState(int backtrackLevel)
{
#ifdef GPU
    int vi = KernelUtils::getTaskIndex();
    if (vi >= 0 and vi < backupsStacks.size)
#else
    for (int vi = 0; vi < backupsStacks.size; vi += 1)
#endif
    {
        if (isDomainChanged(vi))
        {
            representations->minimums[vi] = backupsStacks[vi].minimums.back();
            representations->maximums[vi] = backupsStacks[vi].maximums.back();
            representations->offsets[vi] = backupsStacks[vi].offsets.back();
            representations->versions[vi] = backupsStacks[vi].versions.back();
            representations->bitvectors[vi].copy(&backupsStacks[vi].bitvectors.back());

        }
    }
}

cudaDevice void IntBacktrackStack::clearState(int backtrackLevel)
{
#ifdef GPU
    int vi = KernelUtils::getTaskIndex();
    if (vi >= 0 and vi < backupsStacks.size)
#else
    for (int vi = 0; vi < backupsStacks.size; vi += 1)
#endif
    {
        if (levelsStacks[vi].back() == backtrackLevel)
        {
            backupsStacks[vi].pop();
            levelsStacks[vi].pop_back();
        }
    }
}

cudaDevice bool IntBacktrackStack::isDomainChanged(int variable)
{
    bool changed = backupsStacks[variable].versions.back() != representations->versions[variable];
    return changed;
}
