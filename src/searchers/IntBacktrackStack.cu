#include "hip/hip_runtime.h"
#include <searchers/IntBacktrackStack.h>
#include <utils/KernelUtils.h>

void IntBacktrackStack::initialize(IntDomainsRepresentations* representations)
{
    this->representations = representations;

    backupsStacks.initialize(representations->bitvectors.size);
    backupsStacks.resize(representations->bitvectors.size);

    levelsStacks.initialize(representations->bitvectors.size);
    levelsStacks.resize(representations->bitvectors.size);

    for (int vi = 0; vi < backupsStacks.size; vi += 1)
    {
        backupsStacks[vi].initialize(VECTOR_INITIAL_CAPACITY);

        levelsStacks[vi].initialize();
    }
}

void IntBacktrackStack::deinitialize()
{
    for (int vi = 0; vi < backupsStacks.size; vi += 1)
    {
        backupsStacks[vi].deinitialize();
        levelsStacks[vi].deinitialize();
    }
    backupsStacks.deinitialize();
    levelsStacks.deinitialize();
}

/**
* Saves the current domain representation for each variable in the stack,
* with the given backtrack level (assuming there has been a change).
*/
cudaDevice void IntBacktrackStack::saveState(int backtrackLevel)
{
#ifdef GPU
    int vi = KernelUtils::getTaskIndex();
    if (vi >= 0 and vi < backupsStacks.size)
#else
    for (int vi = 0; vi < backupsStacks.size; vi += 1)
#endif
    {
        if (backtrackLevel == 0 || isDomainChanged(vi))
        {
            int min = representations->minimums[vi];
            int max = representations->maximums[vi];
            int offset = representations->offsets[vi];
            int version = representations->versions[vi];
            Vector<unsigned int>* bitvector = &representations->bitvectors[vi];
            backupsStacks[vi].push(min, max, offset, version, bitvector);

            levelsStacks[vi].push_back(backtrackLevel);
        }
    }
}

/**
* Restores the domain representation with the given backtrack level,
* assuming it is different.
*/
cudaDevice void IntBacktrackStack::restoreState(int backtrackLevel)
{
#ifdef GPU
    int vi = KernelUtils::getTaskIndex();
    if (vi >= 0 and vi < backupsStacks.size)
#else
    for (int vi = 0; vi < backupsStacks.size; vi += 1)
#endif
    {
        if (isDomainChanged(vi))
        {
            representations->minimums[vi] = backupsStacks[vi].minimums.back();
            representations->maximums[vi] = backupsStacks[vi].maximums.back();
            representations->offsets[vi] = backupsStacks[vi].offsets.back();
            representations->versions[vi] = backupsStacks[vi].versions.back();
            representations->bitvectors[vi].copy(&backupsStacks[vi].bitvectors.back());

        }
    }
}

/**
* Clears the state (domain representations) with the given backtrack level
* from the stack.
*/
cudaDevice void IntBacktrackStack::clearState(int backtrackLevel)
{
#ifdef GPU
    int vi = KernelUtils::getTaskIndex();
    if (vi >= 0 and vi < backupsStacks.size)
#else
    for (int vi = 0; vi < backupsStacks.size; vi += 1)
#endif
    {
        if (levelsStacks[vi].back() == backtrackLevel)
        {
            backupsStacks[vi].pop();
            levelsStacks[vi].pop_back();
        }
    }
}

/**
* \return true if "variable" has a different domain (cardinality) in the state 
* with the last backtrack level.
*/
cudaDevice bool IntBacktrackStack::isDomainChanged(int variable)
{
    return backupsStacks[variable].versions.back() != representations->versions[variable];
}
