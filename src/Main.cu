#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <chrono>
#include <random>

#include <utils/Utils.h>
#include <flatzinc/flatzinc.h>
#include <searchers/IntBacktrackSearcher.h>
#include <searchers/IntLNSSearcher.h>
#include <options/Options.h>
#include <wrappers/Wrappers.h>
#include <variables/IntNeighborhood.h>

using namespace std;

int main(int argc, char * argv[])
{
    //-------------------------------------------------------------------------------
    // Start timer
    //-------------------------------------------------------------------------------
    std::chrono::steady_clock::time_point startTime {std::chrono::steady_clock::now()};
    
    IntBacktrackSearcher* backtrackSearcher;
    IntLNSSearcher* LNSSearcher;
    IntSNBSearcher* SNBSearcher;
    //-------------------------------------------------------------------------------
    // Parse command line arguments
    //-------------------------------------------------------------------------------
    Options opts;
    opts.initialize();
    opts.parseOptions(argc, argv);
    //-------------------------------------------------------------------------------
    // Initialize FlatZinc model and printer
    //-------------------------------------------------------------------------------
    FlatZinc::Printer printer;
    FlatZinc::FlatZincModel* fzModel = FlatZinc::parse(opts.inputFile, printer);

    // Max elapsed time in ns
    long long timeout = opts.timeout * 1000000;
    cout << "Timeout: " << opts.timeout << " ms" << endl ;

    // Number of neighborhoods processed in parallel
    int neighborhoodsAmount = 32;
    int neighborhoodsBlocksCount = KernelUtils::getBlockCount(neighborhoodsAmount, DEFAULT_BLOCK_SIZE, true);
    
    int varCount = fzModel->intVariables->count;

    //-------------------------------------------------------------------------------
    // Initialize searcher
    //-------------------------------------------------------------------------------
    if(opts.mode == Options::SearchMode::Backtracking)
    {
        MemUtils::malloc(&backtrackSearcher);
        backtrackSearcher->initialize(fzModel);
    }

    bool* satisfiableModel;
    MemUtils::malloc(&satisfiableModel); // Must be readable by GPU
    *satisfiableModel = true;
    
    //-------------------------------------------------------------------------------
    // Make sure the model is satisfiable, by propagating the constaints. (GPU/CPU)
    //-------------------------------------------------------------------------------
    #ifdef GPU
    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSetLimit(hipLimitMallocHeapSize, HEAP_SIZE));
    #endif
    switch(opts.mode)
    {
        case Options::SearchMode::Backtracking:
            #ifdef GPU
            Wrappers::propagateConstraints<<<1, 1>>>(&backtrackSearcher->
                propagator, satisfiableModel);
            LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
            #else
            *satisfiableModel = backtrackSearcher->propagator.propagateConstraints();
            #endif
            break;

        case Options::SearchMode::LNS:
        case Options::SearchMode::SNBS:
            IntConstraintsPropagator* tempProp;
            MemUtils::malloc(&tempProp);
            tempProp->initialize(fzModel->intVariables, fzModel->intConstraints);
            #ifdef GPU
            Wrappers::propagateConstraints<<<1, 1>>>(tempProp, satisfiableModel);
            LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
            #else
            *satisfiableModel = tempProp->propagateConstraints();
            #endif
            tempProp->deinitialize();
            break;
    }
    //-------------------------------------------------------------------------------
    //-------------------------------------------------------------------------------
    //-------------------------------------------------------------------------------
    // LNS & Co. only: backup original domains after the first propagation
    // Initialize searchers
    //-------------------------------------------------------------------------------
    //-------------------------------------------------------------------------------
    //-------------------------------------------------------------------------------
    if(opts.mode == Options::SearchMode::LNS or opts.mode == Options::SearchMode::SNBS)
    {
        IntDomainsRepresentations* originalDomains;
        MemUtils::malloc(&originalDomains);
        int varCount {fzModel->intVariables->count};
        originalDomains->initialize(varCount);
        IntDomainsRepresentations* intDomRepr  = &fzModel->intVariables->domains.representations;
        for (int vi = 0; vi < varCount; vi += 1)
        {   
            int min = intDomRepr->minimums[vi];
            int max = intDomRepr->maximums[vi];
            int offset = intDomRepr->offsets[vi];
            int version = intDomRepr->versions[vi];
            Vector<unsigned int>* bitvector = &intDomRepr->bitvectors[vi];
            originalDomains->push(min, max, offset, version, bitvector);
        }
        if(opts.mode == Options::SearchMode::LNS)
        {
            MemUtils::malloc(&LNSSearcher);
            LNSSearcher->initialize(fzModel, opts.unassignRate, neighborhoodsAmount, originalDomains);
            for(int j = 0; j < neighborhoodsAmount; j++)
            {
                printf("Neigh %d: (",j);
                for(int i = 0; i < LNSSearcher->neighborhoods[j]->count-1; i++)
                {
                    printf("%d, ", LNSSearcher->neighborhoods[j]->map[i]);
                }
                printf("%d)\n", LNSSearcher->neighborhoods[j]->map[LNSSearcher->neighborhoods[j]->count-1]);
            }
        }
        else
        {
            //~ MemUtils::malloc(&SNBSearcher);
            //~ SNBSearcher->initialize(fzModel,opts.unassignAmount, opts.iterations, , originalDomains);
        }
    }
                
    long long elapsedTime { std::chrono::duration_cast<std::chrono::nanoseconds>(
        std::chrono::steady_clock::now() - startTime).count() };

    if (*satisfiableModel)
    {
        bool* solutionFound;
        MemUtils::malloc(&solutionFound);
        *solutionFound = true;

        unsigned int solutionCount = 0;

        // Check if only the best solution is required
        bool onlyBestSolution = false;
        switch(opts.mode)
        {
            case Options::SearchMode::Backtracking:
                onlyBestSolution = onlyBestSolution or backtrackSearcher->searchType ==
                    IntBacktrackSearcher::SearchType::Maximization;
                onlyBestSolution = onlyBestSolution or backtrackSearcher->searchType ==
                    IntBacktrackSearcher::SearchType::Minimization;
                break;

            case Options::SearchMode::LNS:
                onlyBestSolution = onlyBestSolution or LNSSearcher->searchType ==
                    IntLNSSearcher::SearchType::Maximization;
                onlyBestSolution = onlyBestSolution or LNSSearcher->searchType ==
                    IntLNSSearcher::SearchType::Minimization;
                break;
            
            case Options::SearchMode::SNBS:
                onlyBestSolution = onlyBestSolution or SNBSearcher->searchType ==
                    IntSNBSearcher::SearchType::Maximization;
                onlyBestSolution = onlyBestSolution or SNBSearcher->searchType ==
                    IntSNBSearcher::SearchType::Minimization;
                break;
        }
        onlyBestSolution = onlyBestSolution and opts.solutionsCount == 1;
        std::stringstream bestSolution;
        //-------------------------------------------------------------------------------
        //-------------------------------------------------------------------------------
        // LNS & Co. only: find a first solution
        //-------------------------------------------------------------------------------
        //-------------------------------------------------------------------------------
        if(opts.mode == Options::SearchMode::LNS or opts.mode == Options::SearchMode::SNBS)
        {
            IntDomainsRepresentations* solDomRepr;
            MemUtils::malloc(&solDomRepr);
            int varCount {fzModel->intVariables->count};
            MemUtils::malloc(&backtrackSearcher);
            backtrackSearcher->initialize(fzModel);
            #ifdef GPU
            Wrappers::getNextSolution<<<1, 1>>>(backtrackSearcher, solutionFound, timeout - elapsedTime);
            LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
            #else
            solutionFound = backtrackSearcher->getNextSolution();
            #endif
            //backtrackSearcher->deinitialize(); // FIXME
            assert(solutionFound);
            // Print/store the found solution.
            if (not onlyBestSolution)
            {
                solutionCount += 1;

                printer.print(cout, *fzModel);
                cout << "----------" << endl;
            }
            else
            {
                solutionCount = 1;

                bestSolution.str("");
                printer.print(bestSolution, *fzModel);
            }
            solDomRepr->initialize(varCount);
            IntDomainsRepresentations* intDomRepr  = &fzModel->intVariables->domains.representations;
            for (int vi = 0; vi < varCount; vi += 1)
            {   
                int min = intDomRepr->minimums[vi];
                int max = intDomRepr->maximums[vi];
                int offset = intDomRepr->offsets[vi];
                int version = intDomRepr->versions[vi];
                Vector<unsigned int>* bitvector = &intDomRepr->bitvectors[vi];
                solDomRepr->push(min, max, offset, version, bitvector);
            }
            
            if(opts.mode == Options::SearchMode::LNS)
            {
                LNSSearcher->bestSolution = solDomRepr;
            }
            else
            {
                //SNBSearcher->bestSolution = solDomRepr;
            }
            elapsedTime = std::chrono::duration_cast<std::chrono::nanoseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
            cout << "Initial solution found (" << elapsedTime / 1000000000.0 << "s)." << endl;
            cout << bestSolution.rdbuf();
            cout << "----------" << endl;
        }
        //-------------------------------------------------------------------------------
        /*
        * Find solutions until the search criteria are met.
        * That means finding one/n/all solutions, depending on the user
        * provided arguments.
        */
        //-------------------------------------------------------------------------------
        while (*solutionFound and 
               (solutionCount < opts.solutionsCount or onlyBestSolution) and 
               elapsedTime < timeout
              )
        {
            elapsedTime = std::chrono::duration_cast<std::chrono::nanoseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
            long long searcherTimeout {timeout - elapsedTime};
            //-------------------------------------------------------------------------------
            // Get next solution (GPU/CPU)
            //-------------------------------------------------------------------------------
            switch(opts.mode)
            {
                case Options::SearchMode::Backtracking:
                    #ifdef GPU
                    Wrappers::getNextSolution<<<1, 1>>>(backtrackSearcher, solutionFound, searcherTimeout);
                    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
                    #else
                    solutionFound = backtrackSearcher->getNextSolution(searcherTimeout);
                    #endif
                    break;

                case Options::SearchMode::LNS:
                    cout << "Launching "<< neighborhoodsAmount << " LNS solvers" << endl;
                    #ifdef GPU
                    Wrappers::getNextSolution<<<neighborhoodsBlocksCount, DEFAULT_BLOCK_SIZE>>>
                        (LNSSearcher, searcherTimeout);
                    // Synchronize after resetting
                    #else
                    *solutionFound = LNSSearcher->getNextSolution(searcherTimeout);
                    #endif
                    break;
                
                case Options::SearchMode::SNBS:
                    #ifdef GPU
                    Wrappers::getNextSolution<<<1, 1>>>(SNBSearcher, solutionFound, searcherTimeout);
                    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
                    #else
                    *solutionFound = SNBSearcher->getNextSolution(searcherTimeout);
                    #endif
                    break;
            }
            
            //-------------------------------------------------------------------------------
            //-------------------------------------------------------------------------------
            //-------------------------------------------------------------------------------
            // LNS & Co. only: generate new neighborhoods
            //-------------------------------------------------------------------------------
            //-------------------------------------------------------------------------------
            //-------------------------------------------------------------------------------
            if(opts.mode == Options::SearchMode::LNS /*or opts.mode == Options::SearchMode::SNBS*/)
            {
                // REMEMBER! Accessing unified memory (i.e. almost everything
                // in here) means lots of SIGBUS exceptions!
                // ABSOLUTELY no UM access from host while kernels are running!
                cout << "Generating new neighborhoods in the meantime..." ;
                int randSeed = 1337; // FIXME
                std::mt19937 mt_rand = std::mt19937(randSeed);
                IntVariables* variables = fzModel->intVariables;
                int unassignAmount = varCount * opts.unassignRate;
                if(unassignAmount < 1) unassignAmount = 1;

                Vector<IntNeighborhood*> newNeighs;
                newNeighs.initialize(neighborhoodsAmount);
                
                int* neighVars[neighborhoodsAmount];
                int shuffledVars[varCount];
                
                for(int nbh = 0; nbh < neighborhoodsAmount; nbh += 1)
                {
                    neighVars[nbh] = new int [unassignAmount+1];
                    // Fill variables vector to be shuffled
                    for(int i = 0; i < varCount; i += 1)
                    {
                        //shuffledVars.push_back(i);
                        shuffledVars[i] = i;
                    }
                    
                    // Shuffle (Fisher-Yates/Knuth)
                    for(int i = 0; i < varCount-1; i += 1)
                    {
                        // We want a random variable index (bar the optVariable)
                        std::uniform_int_distribution<int> rand_dist(i, varCount-2);
                        int j{rand_dist(mt_rand)};
                        int tmp{shuffledVars[i]};
                        shuffledVars[i] = shuffledVars[j];
                        shuffledVars[j] = tmp;
                    }
                    // Copy the required subset of the shuffled variables
                    for(int i = 0; i < unassignAmount; i++)
                    {
                        neighVars[nbh][i]=shuffledVars[i];
                    }
                    neighVars[nbh][unassignAmount] = fzModel->optVar();
                }
                
                cout << "done." << endl;
                LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
                
                cout << "Reinitializing searcher..." << endl;
                LNSSearcher->reinitialize(neighVars);
                
                Wrappers::restoreBestSolution<<<LNSSearcher->variablesBlockCount, DEFAULT_BLOCK_SIZE>>>(LNSSearcher);
                LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
            }
            
            //-------------------------------------------------------------------------------
            // Measure time elapsed
            //-------------------------------------------------------------------------------
            elapsedTime = std::chrono::duration_cast<std::chrono::nanoseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
            
            if (*solutionFound)
            {
                // Print/store the found solution.
                if (not onlyBestSolution)
                {
                    solutionCount += 1;

                    printer.print(cout, *fzModel);
                    cout << "----------" << endl;
                }
                else
                {
                    solutionCount = 1;

                    bestSolution.str("");
                    printer.print(bestSolution, *fzModel);
                }
            }
        } // END WHILE
        // Print best solution.
        if(onlyBestSolution)
        {
            cout << bestSolution.rdbuf();
            cout << "----------" << endl;
        }

        if (solutionCount > 0)
        {
            cout << "==========" << endl;
        }
        else
        {
            cout << "=====UNSATISFIABLE=====" << endl;
        }
    }
    else
    {
        cout << "=====UNSATISFIABLE=====" << endl;
    }

    elapsedTime = std::chrono::duration_cast<std::chrono::nanoseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
    cout << "Elapsed time: " << elapsedTime / 1000000000.0 << " s" << endl;
    //-------------------------------------------------------------------------------
    // Print timeout message
    //-------------------------------------------------------------------------------
    if(elapsedTime >= timeout)
    {
        cout << ">>> Timed out! <<<" << endl;
    }
    
    return EXIT_SUCCESS;
}
