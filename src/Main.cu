#include <cstdlib>
#include <iostream>
#include <sstream>
#include <chrono>

#include <utils/Utils.h>
#include <flatzinc/flatzinc.h>
#include <searchers/IntBacktrackSearcher.h>
#include <searchers/IntLNSSearcher.h>
#include <options/Options.h>
#include <wrappers/Wrappers.h>

using namespace std;

int launchBTS(Options* options);
int launchLNS(Options* options);
int launchSNBS(Options* options);

int main(int argc, char * argv[])
{
    // Parse command line arguments
    Options opts;
    opts.initialize();
    opts.parseOptions(argc, argv);

    switch(opts.mode)
    {
        case Options::SearchMode::Backtracking:
            return launchBTS(&opts);

        case Options::SearchMode::LNS:
            return launchLNS(&opts);
        
        case Options::SearchMode::SNBS:
            return launchSNBS(&opts);
    }
}

//-------------------------------------------------------------------
//-------------------------------------------------------------------
//-------------------------------------------------------------------
//-------------------------------------------------------------------
//-------------------------------------------------------------------

int launchSNBS(Options* options)
{
    Options& opts = *options;
    // Initialize FlatZinc model and printer
    FlatZinc::Printer printer;
    FlatZinc::FlatZincModel* fzModel = FlatZinc::parse(opts.inputFile, printer);

    // Initialize searcher
    IntSNBSearcher* SNBSearcher;
    MemUtils::malloc(&SNBSearcher);
    SNBSearcher->initialize(fzModel,opts.unassignAmount, opts.iterations);

    bool* satisfiableModel;
    MemUtils::malloc(&satisfiableModel); // Must be readable by GPU
    *satisfiableModel = true;
    
    // Max elapsed time in ms
    long timeout = opts.timeout;
    cout << "Timeout: " << timeout << endl ;
    std::chrono::steady_clock::time_point startTime {std::chrono::steady_clock::now()};
    
    // Make sure the model is satisfiable, by propagating the constaints. (GPU/CPU)
#ifdef GPU
    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSetLimit(hipLimitMallocHeapSize, HEAP_SIZE));

    Wrappers::propagateConstraints<<<1, 1>>>(&SNBSearcher->propagator,
                                             satisfiableModel);
    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
#else
    *satisfiableModel = SNBSearcher->propagator.propagateConstraints();
#endif
    
    long elapsedTime { std::chrono::duration_cast<std::chrono::milliseconds>(
        std::chrono::steady_clock::now() - startTime).count() };

    if (*satisfiableModel)
    {
        bool* solutionFound;
        MemUtils::malloc(&solutionFound);
        *solutionFound = true;

        unsigned int solutionCount = 0;

        // Check if only the best solution is required
        bool onlyBestSolution = false;
        onlyBestSolution = onlyBestSolution or SNBSearcher->searchType ==
            IntSNBSearcher::SearchType::Maximization;
        onlyBestSolution = onlyBestSolution or SNBSearcher->searchType ==
            IntSNBSearcher::SearchType::Minimization;
        onlyBestSolution = onlyBestSolution and opts.solutionsCount == 1;
        std::stringstream bestSolution;
        
        /*
        * Find solutions until the search criteria are met.
        * That means finding one/n/all solutions, depending on the user
        * provided arguments.
        */
        while (*solutionFound and 
               (solutionCount < opts.solutionsCount or onlyBestSolution) and 
               elapsedTime < timeout
              )
        {
            // Get next solution (GPU/CPU)
            elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
            long searcherTimeout {timeout - elapsedTime};
#ifdef GPU
            Wrappers::getNextSolution<<<1, 1>>>(SNBSearcher, solutionFound, searcherTimeout);
            LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
#else
            //*solutionFound = backtrackSearcher->getNextSolution();
            *solutionFound = SNBSearcher->getNextSolution(searcherTimeout);
#endif
            
            // Measure time
            elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
            //cout << "Solution: " << elapsedTime << endl;
            
            if (*solutionFound)
            {
                // Print/store the found solution.
                if (not onlyBestSolution)
                {
                    solutionCount += 1;

                    printer.print(cout, *fzModel);
                    cout << "----------" << endl;
                }
                else
                {
                    solutionCount = 1;

                    bestSolution.str("");
                    printer.print(bestSolution, *fzModel);
                }
            }
        }

        // Print best solution.
        if(onlyBestSolution)
        {
            cout << bestSolution.rdbuf();
            cout << "----------" << endl;
        }

        if (solutionCount > 0)
        {
            cout << "==========" << endl;
        }
        else
        {
            cout << "=====UNSATISFIABLE=====" << endl;
        }
    }
    else
    {
        cout << "=====UNSATISFIABLE=====" << endl;
    }

    elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
    cout << "Elapsed time: " << elapsedTime << " ms" << endl;
    
    return EXIT_SUCCESS;
}

//-------------------------------------------------------------------
//-------------------------------------------------------------------
//-------------------------------------------------------------------
//-------------------------------------------------------------------
//-------------------------------------------------------------------

int launchLNS(Options* options)
{
    Options& opts = *options;
    // Initialize FlatZinc model and printer
    FlatZinc::Printer printer;
    FlatZinc::FlatZincModel* fzModel = FlatZinc::parse(opts.inputFile, printer);
    // Initialize searcher
    IntLNSSearcher* LNSSearcher;
    MemUtils::malloc(&LNSSearcher);
    LNSSearcher->initialize(fzModel,opts.unassignRate, opts.iterations);

    bool* satisfiableModel;
    MemUtils::malloc(&satisfiableModel); // Must be readable by GPU
    *satisfiableModel = true;
    
    // Max elapsed time in ms
    long timeout = opts.timeout;
    cout << "Timeout: " << timeout << endl ;
    std::chrono::steady_clock::time_point startTime {std::chrono::steady_clock::now()};
    
    // Make sure the model is satisfiable, by propagating the constaints. (GPU/CPU)
#ifdef GPU
    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSetLimit(hipLimitMallocHeapSize, HEAP_SIZE));

    Wrappers::propagateConstraints<<<1, 1>>>(&LNSSearcher->BTSearcher.propagator,
                                             satisfiableModel);
    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
#else
    *satisfiableModel = LNSSearcher->BTSearcher.propagator.propagateConstraints();
#endif
    
    long elapsedTime { std::chrono::duration_cast<std::chrono::milliseconds>(
        std::chrono::steady_clock::now() - startTime).count() };

    if (*satisfiableModel)
    {
        bool* solutionFound;
        MemUtils::malloc(&solutionFound);
        *solutionFound = true;

        unsigned int solutionCount = 0;

        // Check if only the best solution is required
        bool onlyBestSolution = false;
        onlyBestSolution = onlyBestSolution or LNSSearcher->searchType ==
            IntLNSSearcher::SearchType::Maximization;
        onlyBestSolution = onlyBestSolution or LNSSearcher->searchType ==
            IntLNSSearcher::SearchType::Minimization;
        onlyBestSolution = onlyBestSolution and opts.solutionsCount == 1;
        std::stringstream bestSolution;
        
        /*
        * Find solutions until the search criteria are met.
        * That means finding one/n/all solutions, depending on the user
        * provided arguments.
        */
        while (*solutionFound and 
               (solutionCount < opts.solutionsCount or onlyBestSolution) and 
               elapsedTime < timeout
              )
        {
            // Get next solution (GPU/CPU)
            elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
            long searcherTimeout {timeout - elapsedTime};
#ifdef GPU
            Wrappers::getNextSolution<<<1, 1>>>(LNSSearcher, solutionFound, searcherTimeout);
            LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
#else
            *solutionFound = LNSSearcher->getNextSolution(searcherTimeout);
#endif
            
            // Measure time
            elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
            //cout << "Solution: " << elapsedTime << endl;
            
            if (*solutionFound)
            {
                // Print/store the found solution.
                if (not onlyBestSolution)
                {
                    solutionCount += 1;

                    printer.print(cout, *fzModel);
                    cout << "----------" << endl;
                }
                else
                {
                    solutionCount = 1;

                    bestSolution.str("");
                    printer.print(bestSolution, *fzModel);
                }
            }
        }

        // Print best solution.
        if(onlyBestSolution)
        {
            cout << bestSolution.rdbuf();
            cout << "----------" << endl;
        }

        if (solutionCount > 0)
        {
            cout << "==========" << endl;
        }
        else
        {
            cout << "=====UNSATISFIABLE=====" << endl;
        }
    }
    else
    {
        cout << "=====UNSATISFIABLE=====" << endl;
    }

    elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
    cout << "Elapsed time: " << elapsedTime << " ms" << endl;
    
    return EXIT_SUCCESS;
}

//-------------------------------------------------------------------
//-------------------------------------------------------------------
//-------------------------------------------------------------------
//-------------------------------------------------------------------
//-------------------------------------------------------------------

int launchBTS(Options* options)
{
    Options& opts = *options;
    // Initialize FlatZinc model and printer
    FlatZinc::Printer printer;
    FlatZinc::FlatZincModel* fzModel = FlatZinc::parse(opts.inputFile, printer);
    // Initialize backtrack searcher
    IntBacktrackSearcher* backtrackSearcher;
    MemUtils::malloc(&backtrackSearcher);
    backtrackSearcher->initialize(fzModel);

    bool* satisfiableModel;
    MemUtils::malloc(&satisfiableModel); // Must be readable by GPU
    *satisfiableModel = true;
    
    // Max elapsed time in ms
    long timeout = opts.timeout;
    cout << "Timeout: " << timeout << endl ;
    std::chrono::steady_clock::time_point startTime {std::chrono::steady_clock::now()};
    
    // Make sure the model is satisfiable, by propagating the constaints. (GPU/CPU)
#ifdef GPU
    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSetLimit(hipLimitMallocHeapSize, HEAP_SIZE));

    Wrappers::propagateConstraints<<<1, 1>>>(&backtrackSearcher->propagator, satisfiableModel);
    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
#else
    *satisfiableModel = backtrackSearcher->propagator.propagateConstraints();
#endif
    
    long elapsedTime { std::chrono::duration_cast<std::chrono::milliseconds>(
        std::chrono::steady_clock::now() - startTime).count() };

    if (*satisfiableModel)
    {
        bool* solutionFound;
        MemUtils::malloc(&solutionFound);
        *solutionFound = true;

        unsigned int solutionCount = 0;

        // Check if only the best solution is required
        bool onlyBestSolution = false;
        onlyBestSolution = onlyBestSolution or backtrackSearcher->searchType == IntBacktrackSearcher::SearchType::Maximization;
        onlyBestSolution = onlyBestSolution or backtrackSearcher->searchType == IntBacktrackSearcher::SearchType::Minimization;
        onlyBestSolution = onlyBestSolution and opts.solutionsCount == 1;
        std::stringstream bestSolution;
        
        /*
        * Find solutions until the search criteria are met.
        * That means finding one/n/all solutions, depending on the user
        * provided arguments.
        */
        while (*solutionFound and 
               (solutionCount < opts.solutionsCount or onlyBestSolution) and 
               elapsedTime < timeout
              )
        {
            // Get next solution (GPU/CPU)
            elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
            long searcherTimeout {timeout - elapsedTime};
#ifdef GPU
            Wrappers::getNextSolution<<<1, 1>>>(backtrackSearcher, solutionFound); // timeout
            LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
#else
            *solutionFound = backtrackSearcher->getNextSolution(); // timeout
#endif
            
            // Measure time
            elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
            //cout << "Solution: " << elapsedTime << endl;
            
            if (*solutionFound)
            {
                // Print/store the found solution.
                if (not onlyBestSolution)
                {
                    solutionCount += 1;

                    printer.print(cout, *fzModel);
                    cout << "----------" << endl;
                }
                else
                {
                    solutionCount = 1;

                    bestSolution.str("");
                    printer.print(bestSolution, *fzModel);
                }
            }
        }

        // Print best solution.
        if(onlyBestSolution)
        {
            cout << bestSolution.rdbuf();
            cout << "----------" << endl;
        }

        if (solutionCount > 0)
        {
            cout << "==========" << endl;
        }
        else
        {
            cout << "=====UNSATISFIABLE=====" << endl;
        }
    }
    else
    {
        cout << "=====UNSATISFIABLE=====" << endl;
    }

    elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
    cout << "Elapsed time: " << elapsedTime << " ms" << endl;
    
    return EXIT_SUCCESS;
}
