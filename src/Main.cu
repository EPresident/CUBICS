#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <chrono>
#include <random>

#include <utils/Utils.h>
#include <flatzinc/flatzinc.h>
#include <searchers/IntBacktrackSearcher.h>
#include <searchers/IntLNSSearcher.h>
#include <options/Options.h>
#include <wrappers/Wrappers.h>
#include <variables/IntNeighborhood.h>

using namespace std;

int main(int argc, char * argv[])
{
    //-------------------------------------------------------------------------------
    // Start timer
    //-------------------------------------------------------------------------------
    std::chrono::steady_clock::time_point startTime {std::chrono::steady_clock::now()};
    
    IntBacktrackSearcher* backtrackSearcher;
    IntLNSSearcher* LNSSearcher;
    IntSNBSearcher* SNBSearcher;
    //-------------------------------------------------------------------------------
    // Parse command line arguments
    //-------------------------------------------------------------------------------
    Options opts;
    opts.initialize();
    opts.parseOptions(argc, argv);
    //-------------------------------------------------------------------------------
    // Initialize FlatZinc model and printer
    //-------------------------------------------------------------------------------
    FlatZinc::Printer printer;
    FlatZinc::FlatZincModel* fzModel = FlatZinc::parse(opts.inputFile, printer);

    // Max elapsed time in ns
    long long timeout = opts.timeout * 1000000;
    cout << "Timeout: " << opts.timeout << " ms" << endl ;

    // Number of neighborhoods processed in parallel
    int neighborhoodsAmount = 2;
    int neighborhoodsBlocksCount = KernelUtils::getBlockCount(neighborhoodsAmount, DEFAULT_BLOCK_SIZE, true);

    //-------------------------------------------------------------------------------
    // Initialize searcher
    //-------------------------------------------------------------------------------
    if(opts.mode == Options::SearchMode::Backtracking)
    {
        MemUtils::malloc(&backtrackSearcher);
        backtrackSearcher->initialize(fzModel);
    }

    bool* satisfiableModel;
    MemUtils::malloc(&satisfiableModel); // Must be readable by GPU
    *satisfiableModel = true;
    
    //-------------------------------------------------------------------------------
    // Make sure the model is satisfiable, by propagating the constaints. (GPU/CPU)
    //-------------------------------------------------------------------------------
    #ifdef GPU
    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSetLimit(hipLimitMallocHeapSize, HEAP_SIZE));
    #endif
    switch(opts.mode)
    {
        case Options::SearchMode::Backtracking:
            #ifdef GPU
            Wrappers::propagateConstraints<<<1, 1>>>(&backtrackSearcher->
                propagator, satisfiableModel);
            LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
            #else
            *satisfiableModel = SNBSearcher->propagator.propagateConstraints();
            #endif
            break;

        case Options::SearchMode::LNS:
        case Options::SearchMode::SNBS:
            IntConstraintsPropagator* tempProp;
            MemUtils::malloc(&tempProp);
            tempProp->initialize(fzModel->intVariables, fzModel->intConstraints);
            #ifdef GPU
            Wrappers::propagateConstraints<<<1, 1>>>(tempProp, satisfiableModel);
            LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
            #else
            *satisfiableModel = tempProp->propagateConstraints();
            #endif
            tempProp->deinitialize();
            break;
    }
    //-------------------------------------------------------------------------------
    //-------------------------------------------------------------------------------
    //-------------------------------------------------------------------------------
    // LNS & Co. only: backup original domains after the first propagation
    // Initialize searchers
    //-------------------------------------------------------------------------------
    //-------------------------------------------------------------------------------
    //-------------------------------------------------------------------------------
    if(opts.mode == Options::SearchMode::LNS or opts.mode == Options::SearchMode::SNBS)
    {
        IntDomainsRepresentations* originalDomains;
        MemUtils::malloc(&originalDomains);
        int varCount {fzModel->intVariables->count};
        originalDomains->initialize(varCount);
        IntDomainsRepresentations* intDomRepr  = &fzModel->intVariables->domains.representations;
        for (int vi = 0; vi < varCount; vi += 1)
        {   
            int min = intDomRepr->minimums[vi];
            int max = intDomRepr->maximums[vi];
            int offset = intDomRepr->offsets[vi];
            int version = intDomRepr->versions[vi];
            Vector<unsigned int>* bitvector = &intDomRepr->bitvectors[vi];
            originalDomains->push(min, max, offset, version, bitvector);
        }
        if(opts.mode == Options::SearchMode::LNS)
        {
            MemUtils::malloc(&LNSSearcher);
            LNSSearcher->initialize(fzModel, opts.unassignRate, neighborhoodsAmount, originalDomains);
            for(int j = 0; j < neighborhoodsAmount; j++)
            {
                printf("Neigh %d: (",j);
                for(int i = 0; i < LNSSearcher->neighborhoods[j]->count-1; i++)
                {
                    printf("%d, ", LNSSearcher->neighborhoods[j]->map[i]);
                }
                printf("%d)\n", LNSSearcher->neighborhoods[j]->map[LNSSearcher->neighborhoods[j]->count-1]);
            }
        }
        else
        {
            //~ MemUtils::malloc(&SNBSearcher);
            //~ SNBSearcher->initialize(fzModel,opts.unassignAmount, opts.iterations, , originalDomains);
        }
    }
    
    //-------------------------------------------------------------------------------
    //-------------------------------------------------------------------------------
    //-------------------------------------------------------------------------------
    // LNS & Co. only: generate neighborhoods with Fisher-Yates
    // Initialize neighborhoods
    //-------------------------------------------------------------------------------
    //-------------------------------------------------------------------------------
    //-------------------------------------------------------------------------------
    //~ if(opts.mode == Options::SearchMode::LNS or opts.mode == Options::SearchMode::SNBS)
    //~ {
        //~ Vector<IntNeighborhood*> neighborhoods;
        //~ long randSeed = 1273916546123835; // Arbitrary seed, FIXME
        //~ std::mt19937 mt_rand = std::mt19937(randSeed);
        //~ int optVariable = fzModel->optVar();
        //~ int unassignAmount = opts.unassignAmount;
        //~ if(opts.mode == Options::SearchMode::LNS) unassignAmount = LNSSearcher->unassignAmount;
        //~ neighborhoods.initialize(neighborhoodsAmount);
        //~ Vector<int> neighVars;
        //~ neighVars.initialize(unassignAmount+1);
        //~ Vector<int> shuffledVars;
        //~ shuffledVars.initialize(fzModel->intVariables->count);
        
        //~ for(int nbh = 0; nbh < neighborhoodsAmount; nbh += 1)
        //~ {
            //~ // Fill variables vector to be shuffled
            //~ for(int i = 0; i < fzModel->intVariables->count; i += 1)
            //~ {
                //~ shuffledVars.push_back(i);
            //~ }
            
            //~ // Shuffle (Fisher-Yates/Knuth)
            //~ for(int i = 0; i < fzModel->intVariables->count-1; i += 1)
            //~ {
                //~ // We want a random variable index (bar the optVariable)
                //~ std::uniform_int_distribution<int> rand_dist(i, fzModel->intVariables->count-2);
                //~ int j{rand_dist(mt_rand)};
                //~ int tmp{shuffledVars[i]};
                //~ shuffledVars[i] = shuffledVars[j];
                //~ shuffledVars[j] = tmp;
            //~ }
            //~ // Copy the required subset of the shuffled variables
            //~ for(int i = 0; i < unassignAmount; i++)
            //~ {
                //~ neighVars.push_back(shuffledVars[i]);
            //~ }
            //~ neighVars.push_back(optVariable);
            //~ // Init neighborhood
            //~ IntNeighborhood newNeigh;
            //~ if(opts.mode == Options::SearchMode::LNS)
            //~ {
                //~ newNeigh.initialize(&neighVars, LNSSearcher->originalDomains);
            //~ }
            //~ else
            //~ {
                //~ //newNeigh.initialize(&neighVars, SNBSearcher->originalDomains);
            //~ }
            //~ neighborhoods.push_back(&newNeigh);
            //~ // Clear vectors for reuse
            //~ shuffledVars.clear();
            //~ neighVars.clear();
        //~ }
        //~ neighVars.deinitialize();
        //~ shuffledVars.deinitialize();
        //~ if(opts.mode == Options::SearchMode::LNS)
        //~ {
            //~ LNSSearcher->neighborhoods = &neighborhoods;neighborhoodsBlocksCount
        //~ }
        //~ else
        //~ {
            //~ //SNBSearcher->originalDomains = originalDomains;
        //~ }
    //~ }
    //---------------------------------------------------------------------
                
    long long elapsedTime { std::chrono::duration_cast<std::chrono::nanoseconds>(
        std::chrono::steady_clock::now() - startTime).count() };

    if (*satisfiableModel)
    {
        bool* solutionFound;
        MemUtils::malloc(&solutionFound);
        *solutionFound = true;

        unsigned int solutionCount = 0;

        // Check if only the best solution is required
        bool onlyBestSolution = false;
        switch(opts.mode)
        {
            case Options::SearchMode::Backtracking:
                onlyBestSolution = onlyBestSolution or backtrackSearcher->searchType ==
                    IntBacktrackSearcher::SearchType::Maximization;
                onlyBestSolution = onlyBestSolution or backtrackSearcher->searchType ==
                    IntBacktrackSearcher::SearchType::Minimization;
                break;

            case Options::SearchMode::LNS:
                onlyBestSolution = onlyBestSolution or LNSSearcher->searchType ==
                    IntLNSSearcher::SearchType::Maximization;
                onlyBestSolution = onlyBestSolution or LNSSearcher->searchType ==
                    IntLNSSearcher::SearchType::Minimization;
                break;
            
            case Options::SearchMode::SNBS:
                onlyBestSolution = onlyBestSolution or SNBSearcher->searchType ==
                    IntSNBSearcher::SearchType::Maximization;
                onlyBestSolution = onlyBestSolution or SNBSearcher->searchType ==
                    IntSNBSearcher::SearchType::Minimization;
                break;
        }
        onlyBestSolution = onlyBestSolution and opts.solutionsCount == 1;
        std::stringstream bestSolution;
        //-------------------------------------------------------------------------------
        //-------------------------------------------------------------------------------
        // LNS & Co. only: find a first solution
        //-------------------------------------------------------------------------------
        //-------------------------------------------------------------------------------
        if(opts.mode == Options::SearchMode::LNS or opts.mode == Options::SearchMode::SNBS)
        {
            IntDomainsRepresentations* solDomRepr;
            MemUtils::malloc(&solDomRepr);
            int varCount {fzModel->intVariables->count};
            MemUtils::malloc(&backtrackSearcher);
            backtrackSearcher->initialize(fzModel);
            #ifdef GPU
            Wrappers::getNextSolution<<<1, 1>>>(backtrackSearcher, solutionFound, timeout - elapsedTime);
            LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
            #else
            solutionFound = backtrackSearcher->getNextSolution();
            #endif
            //backtrackSearcher->deinitialize(); // FIXME
            assert(solutionFound);
            // Print/store the found solution.
            if (not onlyBestSolution)
            {
                solutionCount += 1;

                printer.print(cout, *fzModel);
                cout << "----------" << endl;
            }
            else
            {
                solutionCount = 1;

                bestSolution.str("");
                printer.print(bestSolution, *fzModel);
            }
            solDomRepr->initialize(varCount);
            IntDomainsRepresentations* intDomRepr  = &fzModel->intVariables->domains.representations;
            for (int vi = 0; vi < varCount; vi += 1)
            {   
                int min = intDomRepr->minimums[vi];
                int max = intDomRepr->maximums[vi];
                int offset = intDomRepr->offsets[vi];
                int version = intDomRepr->versions[vi];
                Vector<unsigned int>* bitvector = &intDomRepr->bitvectors[vi];
                solDomRepr->push(min, max, offset, version, bitvector);
            }
            if(opts.mode == Options::SearchMode::LNS)
            {
                LNSSearcher->bestSolution = solDomRepr;
            }
            else
            {
                //SNBSearcher->bestSolution = solDomRepr;
            }
            elapsedTime = std::chrono::duration_cast<std::chrono::nanoseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
        }
        //-------------------------------------------------------------------------------
        /*
        * Find solutions until the search criteria are met.
        * That means finding one/n/all solutions, depending on the user
        * provided arguments.
        */
        //-------------------------------------------------------------------------------
        while (*solutionFound and 
               (solutionCount < opts.solutionsCount or onlyBestSolution) and 
               elapsedTime < timeout
              )
        {
            elapsedTime = std::chrono::duration_cast<std::chrono::nanoseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
            long long searcherTimeout {timeout - elapsedTime};
            //-------------------------------------------------------------------------------
            // Get next solution (GPU/CPU)
            //-------------------------------------------------------------------------------
            switch(opts.mode)
            {
                case Options::SearchMode::Backtracking:
                    #ifdef GPU
                    Wrappers::getNextSolution<<<1, 1>>>(backtrackSearcher, solutionFound, searcherTimeout);
                    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
                    #else
                    solutionFound = backtrackSearcher->getNextSolution(searcherTimeout);
                    #endif
                    break;

                case Options::SearchMode::LNS:
                    #ifdef GPU
                    Wrappers::getNextSolution<<<neighborhoodsBlocksCount, DEFAULT_BLOCK_SIZE>>>
                        (LNSSearcher, searcherTimeout);
                    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
                    Wrappers::restoreBestSolution<<<LNSSearcher->variablesBlockCount, DEFAULT_BLOCK_SIZE>>>
                        (LNSSearcher);
                    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
                    #else
                    *solutionFound = LNSSearcher->getNextSolution(searcherTimeout);
                    #endif
                    break;
                
                case Options::SearchMode::SNBS:
                    #ifdef GPU
                    Wrappers::getNextSolution<<<1, 1>>>(SNBSearcher, solutionFound, searcherTimeout);
                    LogUtils::cudaAssert(__PRETTY_FUNCTION__, hipDeviceSynchronize());
                    #else
                    *solutionFound = SNBSearcher->getNextSolution(searcherTimeout);
                    #endif
                    break;
            }
            
            //-------------------------------------------------------------------------------
            // Measure time elapsed
            //-------------------------------------------------------------------------------
            elapsedTime = std::chrono::duration_cast<std::chrono::nanoseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
            
            if (*solutionFound)
            {
                // Print/store the found solution.
                if (not onlyBestSolution)
                {
                    solutionCount += 1;

                    printer.print(cout, *fzModel);
                    cout << "----------" << endl;
                }
                else
                {
                    solutionCount = 1;

                    bestSolution.str("");
                    printer.print(bestSolution, *fzModel);
                }
            }
            if(opts.mode == Options::SearchMode::LNS) break;
        }
        // Print best solution.
        if(onlyBestSolution)
        {
            cout << bestSolution.rdbuf();
            cout << "----------" << endl;
        }

        if (solutionCount > 0)
        {
            cout << "==========" << endl;
        }
        else
        {
            cout << "=====UNSATISFIABLE=====" << endl;
        }
    }
    else
    {
        cout << "=====UNSATISFIABLE=====" << endl;
    }

    elapsedTime = std::chrono::duration_cast<std::chrono::nanoseconds>(
                            std::chrono::steady_clock::now() - startTime).count();
    cout << "Elapsed time: " << elapsedTime / 1000000000.0 << " s" << endl;
    //-------------------------------------------------------------------------------
    // Print timeout message
    //-------------------------------------------------------------------------------
    if(elapsedTime >= timeout)
    {
        cout << ">>> Timed out! <<<" << endl;
    }
    
    return EXIT_SUCCESS;
}
