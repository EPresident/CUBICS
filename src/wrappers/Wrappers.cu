
#include <hip/hip_runtime.h>
#ifdef GPU
#include <wrappers/Wrappers.h>

//Integer backtracking stack
cudaGlobal void Wrappers::saveState(IntBacktrackStack* backtrackStack, int backtrackingLevel)
{
    backtrackStack->saveState(backtrackingLevel);
}

cudaGlobal void Wrappers::restoreState(IntBacktrackStack* backtrackStack, int backtrackingLevel)
{
    backtrackStack->restoreState(backtrackingLevel);
}

cudaGlobal void Wrappers::clearState(IntBacktrackStack* backtrackStack, int backtrackingLevel)
{
    backtrackStack->clearState(backtrackingLevel);
}

//Integer backtracking searcher
cudaGlobal void Wrappers::getNextSolution(IntBacktrackSearcher* backtrackSearcher, bool* solutionFound)
{
    *solutionFound = backtrackSearcher->getNextSolution();
}

//Integer LNS searcher
cudaGlobal void Wrappers::getNextSolution(IntLNSSearcher* LNSSearcher, bool* solutionFound)
{
    *solutionFound = LNSSearcher->getNextSolution();
}

//Integer constraints propagator
cudaGlobal void Wrappers::propagateConstraints(IntConstraintsPropagator* propagator, bool* satisfiableModel)
{
    *satisfiableModel = propagator->propagateConstraints();
}

cudaGlobal void Wrappers::setConstraintsToPropagate(IntConstraintsPropagator* propagator)
{
    propagator->setConstraintsToPropagate();
}

cudaGlobal void Wrappers::collectActions(IntConstraintsPropagator* propagator)
{
    propagator->collectActions();
}

cudaGlobal void Wrappers::clearDomainsEvents(IntConstraintsPropagator* propagator)
{
    propagator->clearDomainsEvents();
}

cudaGlobal void Wrappers::updateDomains(IntConstraintsPropagator* propagator)
{
    propagator->updateDomains();
}

cudaGlobal void Wrappers::clearConstraintsToPropagate(IntConstraintsPropagator* propagator)
{
    propagator->clearConstraintsToPropagate();
}

cudaGlobal void Wrappers::checkEmptyDomains(IntConstraintsPropagator* propagator)
{
    propagator->checkEmptyDomains();
}

cudaGlobal void Wrappers::checkSatisfiedConstraints(IntConstraintsPropagator* propagator)
{
    propagator->checkSatisfiedConstraints();
}
#endif 
