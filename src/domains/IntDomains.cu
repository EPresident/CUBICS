#include "hip/hip_runtime.h"
#include <domains/IntDomains.h>
#include <cassert>

void IntDomains::initialize(int count)
{
    events.initialize(count);

    representations.initialize(count);
    actions.initialize(count);
}

void IntDomains::deinitialize()
{
    events.deinitialize();

    representations.deinitialize();
    actions.deinitialize();
}

/// Add a new domain (for a new variable), ranging from "min" to "max".
void IntDomains::push(int min, int max)
{
    events.push_back(EventTypes::Changed);

    representations.push(min, max);
    actions.push();
}

cudaDevice bool IntDomains::isEmpty(int index, IntNeighborhood* nbh, int reprIdx)
{
    if(reprIdx > 0)
    {
        return nbh->neighRepr.isEmpty(reprIdx);
    }
    if(nbh->isNeighbor(index))
    {
        return nbh->neighRepr.isEmpty(nbh->getRepresentationIndex(index));
    }
    else
    {
        return IntDomains::isEmpty(index);
    }
}

cudaDevice bool IntDomains::isSingleton(int index, IntNeighborhood* nbh, int reprIdx)
{
    if(reprIdx > 0)
    {
        return nbh->neighRepr.isSingleton(reprIdx);
    }
    if(nbh->isNeighbor(index))
    {
        return nbh->neighRepr.isSingleton(nbh->getRepresentationIndex(index));
    }
    else
    {
        return IntDomains::isSingleton(index);
    }
}

cudaDevice unsigned int IntDomains::getApproximateCardinality(int index, IntNeighborhood* nbh, int reprIdx)
{
    if(reprIdx > 0)
    {
        return nbh->neighRepr.getApproximateCardinality(reprIdx);
    }
    if(nbh->isNeighbor(index))
    {
        return nbh->neighRepr.getApproximateCardinality(nbh->getRepresentationIndex(index));
    }
    else
    {
        return IntDomains::getApproximateCardinality(index);
    }
}

cudaDevice int IntDomains::getMin(int index, IntNeighborhood* nbh, int reprIdx)
{
    if(reprIdx > 0)
    {
        return nbh->neighRepr.minimums[reprIdx];
    }
    if(nbh->isNeighbor(index))
    {
        return nbh->neighRepr.minimums[nbh->getRepresentationIndex(index)];
    }
    else
    {
        return IntDomains::getMin(index);
    }
}

cudaDevice int IntDomains::getMax(int index, IntNeighborhood* nbh, int reprIdx)
{
    if(reprIdx > 0)
    {
        return nbh->neighRepr.maximums[reprIdx];
    }
    if(nbh->isNeighbor(index))
    {
        return nbh->neighRepr.maximums[nbh->getRepresentationIndex(index)];
    }
    else
    {
        return IntDomains::getMax(index);
    }
}

/// Reduce the domain on the "index"-th variable to "value" (singleton).
cudaDevice void IntDomains::fixValue(int index, int value)
{
    assert(representations.contain(index, value));

    representations.keepOnly(index, value);
    events[index] = EventTypes::Changed;
}

cudaDevice void IntDomains::fixValue(int index, int value, IntNeighborhood* nbh)
{
    #ifndef NDEBUG
    assert(nbh->isNeighbor(index));
    #endif
    int ridx {nbh->getRepresentationIndex(index)};
    nbh->neighRepr.keepOnly(ridx, value);
    nbh->events[ridx] = EventTypes::Changed;

}

/**
* Perform the domain reduction actions pertaining the "index"-th 
* variable/domain.
* Values outside the bounds and inside the "remove list" are dropped.
*/
cudaDevice void IntDomains::updateDomain(int index)
{
    unsigned int previousVersion = representations.versions[index];

    // Shave off any value outside the bounds
    representations.removeAnyGreaterThan(index, actions.upperbounds[index]);
    representations.removeAnyLesserThan(index, actions.lowerbounds[index]);

    // Remove single elements
    for (int ei = 0; ei < actions.elementsToRemove[index].size; ei += 1)
    {
        representations.remove(index, actions.elementsToRemove[index][ei]);
    }

    // Remove the action after it's been perforned
    actions.clear(index);

    // Push the appropriate events
    if (previousVersion != representations.versions[index])
    {
        events[index] = EventTypes::Changed;
    }
    else
    {
        events[index] = EventTypes::None;
    }
}

cudaDevice void IntDomains::updateDomain(int index, IntNeighborhood* nbh, int reprIdx)
{
    #ifndef NDEBUG
    assert(nbh->isNeighbor(index));
    #endif
    if(reprIdx < 0)
    {
        reprIdx = nbh->getRepresentationIndex(index);
    }
    
    unsigned int previousVersion = nbh->neighRepr.versions[reprIdx];

    // Shave off any value outside the bounds
    nbh->neighRepr.removeAnyGreaterThan(reprIdx, nbh->neighActions.upperbounds[reprIdx]);
    nbh->neighRepr.removeAnyLesserThan(reprIdx, nbh->neighActions.lowerbounds[reprIdx]);

    // Remove single elements
    for (int ei = 0; ei < nbh->neighActions.elementsToRemove[reprIdx].size; ei += 1)
    {
        nbh->neighRepr.remove(reprIdx, nbh->neighActions.elementsToRemove[reprIdx][ei]);
    }

    // Remove the action after it's been perforned
    nbh->neighActions.clear(reprIdx);

    // Push the appropriate events
    if (previousVersion != nbh->neighRepr.versions[reprIdx])
    {
        nbh->events[reprIdx] = EventTypes::Changed;
    }
    else
    {
        nbh->events[reprIdx] = EventTypes::None;
    }
}
