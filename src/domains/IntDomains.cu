#include "hip/hip_runtime.h"
#include <domains/IntDomains.h>

void IntDomains::initialize(int count)
{
    events.initialize(count);

    representations.initialize(count);
    actions.initialize(count);
}

void IntDomains::deinitialize()
{
    events.deinitialize();

    representations.deinitialize();
    actions.deinitialize();
}

void IntDomains::push(int min, int max)
{
    events.push_back(EventTypes::Changed);

    representations.push(min, max);
    actions.push();
}

void IntDomains::fixValue(int index, int value)
{
    assert(representations.contain(index, value));

    representations.keepOnly(index, value);
    events[index] = EventTypes::Changed;
}

void IntDomains::updateDomain(int index)
{
    unsigned int previousVersion = representations.versions[index];

    representations.removeAnyGreaterThan(index, actions.upperbounds[index]);

    representations.removeAnyLesserThan(index, actions.lowerbounds[index]);

    for (int ei = 0; ei < actions.elementsToRemove[index].size; ei += 1)
    {
        representations.remove(index, actions.elementsToRemove[index][ei]);
    }

    actions.clear(index);

    if (previousVersion != representations.versions[index])
    {
        events[index] = EventTypes::Changed;
    }
    else
    {
        events[index] = EventTypes::None;
    }
}
