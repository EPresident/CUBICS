#include "hip/hip_runtime.h"
#include <algorithm>

#include <domains/IntDomainsActions.h>
#include <utils/Utils.h>

void IntDomainsActions::initialize(int count)
{
    elementsToRemove.initialize(count);

    lowerbounds.initialize(count);
    upperbounds.initialize(count);
}

void IntDomainsActions::deinitialize()
{
    for (int i = 0; i < elementsToRemove.size; i += 1)
    {
        elementsToRemove[i].deinitialize();
    }
    elementsToRemove.deinitialize();

    lowerbounds.deinitialize();
    upperbounds.deinitialize();
}

void IntDomainsActions::push()
{
    elementsToRemove.resize_by_one();
    elementsToRemove.back().initialize();

    lowerbounds.push_back(INT_MIN);
    upperbounds.push_back(INT_MAX);
}

cudaDevice void IntDomainsActions::clear(int index)
{
    elementsToRemove[index].clear();

    lowerbounds[index] = INT_MIN;
    upperbounds[index] = INT_MAX;
}

cudaDevice void IntDomainsActions::removeElement(int index, int val)
{
    if (lowerbounds[index] <= val and val <= upperbounds[index])
    {
        elementsToRemove[index].push_back(val);
    }
}

cudaDevice void IntDomainsActions::removeAnyGreaterThan(int index, int val)
{
    upperbounds[index] = std::min(val, upperbounds[index]);
}

cudaDevice void IntDomainsActions::removeAnyLesserThan(int index, int val)
{
    lowerbounds[index] = std::max(val, lowerbounds[index]);
}
