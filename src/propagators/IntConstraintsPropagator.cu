#include <propagators/IntConstraintsPropagator.h>
#include <utils/Utils.h>
#include <wrappers/Wrappers.h>

void IntConstraintsPropagator::initialize(IntVariables* variables, IntConstraints* constraints)
{
    this->variables = variables;
    this->constraints = constraints;

    constraintToPropagate.initialize(constraints->count);
    constraintToPropagate.resize(constraints->count);
    clearConstraintsToPropagate();

#ifdef GPU
    constraintsBlockCountDivergence = KernelUtils::getBlockCount(constraints->count, DEFAULT_BLOCK_SIZE, true);
    constraintsBlockCount = KernelUtils::getBlockCount(constraints->count, DEFAULT_BLOCK_SIZE);
    variablesBlockCount = KernelUtils::getBlockCount(variables->count, DEFAULT_BLOCK_SIZE);
#endif
}

void IntConstraintsPropagator::deinitialize()
{
    constraintToPropagate.deinitialize();
}

cudaDevice bool IntConstraintsPropagator::propagateConstraints()
{
    someEmptyDomain = false;
    someConstraintsToPropagate = false;
#ifdef GPU
    Wrappers::setConstraintsToPropagate<<<constraintsBlockCount, DEFAULT_BLOCK_SIZE>>>(this);
    hipDeviceSynchronize();
#else
    setConstraintsToPropagate();
#endif

    while (someConstraintsToPropagate and (not someEmptyDomain))
    {
#ifdef GPU
        Wrappers::collectActions<<<constraintsBlockCountDivergence, DEFAULT_BLOCK_SIZE>>>(this);
        hipDeviceSynchronize();
#else
        collectActions();
#endif

#ifdef GPU
        Wrappers::clearDomainsEvents<<<variablesBlockCount, DEFAULT_BLOCK_SIZE>>>(this);
        hipDeviceSynchronize();
#else
        clearDomainsEvents();
#endif

#ifdef GPU
        Wrappers::updateDomains<<<variablesBlockCount, DEFAULT_BLOCK_SIZE>>>(this);
        hipDeviceSynchronize();
#else
        updateDomains();
#endif

#ifdef GPU
        Wrappers::clearConstraintsToPropagate<<<constraintsBlockCount, DEFAULT_BLOCK_SIZE>>>(this);
        hipDeviceSynchronize();
#else
        clearConstraintsToPropagate();
#endif

        someEmptyDomain = false;
#ifdef GPU
        Wrappers::checkEmptyDomains<<<variablesBlockCount, DEFAULT_BLOCK_SIZE>>>(this);
        hipDeviceSynchronize();
#else
        checkEmptyDomains();
#endif

        if (not someEmptyDomain)
        {
            someConstraintsToPropagate = false;
#ifdef GPU
            Wrappers::setConstraintsToPropagate<<<constraintsBlockCount, DEFAULT_BLOCK_SIZE>>>(this);
            hipDeviceSynchronize();
#else
            setConstraintsToPropagate();
#endif
        }
    }

    return (not someEmptyDomain);
}

cudaDevice void IntConstraintsPropagator::setConstraintsToPropagate()
{
#ifdef GPU
    int ci = KernelUtils::getTaskIndex();
    if (ci >= 0 and ci < constraints->count)
#else
    for (int ci = 0; ci < constraints->count; ci += 1)
#endif
    {
        for (int vi = 0; vi < constraints->variables[ci].size; vi += 1)
        {
            int event = variables->domains.events[constraints->variables[ci][vi]];

            if (event == IntDomains::EventTypes::Changed)
            {
                constraintToPropagate[ci] = true;
                someConstraintsToPropagate = true;
            }
        }
    }
}

cudaDevice void IntConstraintsPropagator::collectActions()
{
#ifdef GPU
    int ci = KernelUtils::getTaskIndex(true);
    if (ci >= 0 and ci < constraints->count)
#else
    for (int ci = 0; ci < constraints->count; ci += 1)
#endif
    {
        if (constraintToPropagate[ci])
        {
            constraints->propagate(ci, variables);
            constraintToPropagate[ci] = false;
        }
    }
}

cudaDevice void IntConstraintsPropagator::clearDomainsEvents()
{
#ifdef GPU
    int vi = KernelUtils::getTaskIndex();
    if (vi >= 0 and vi < variables->count)
#else
    for (int vi = 0; vi < variables->count; vi += 1)
#endif
    {
        variables->domains.events[vi] = IntDomains::EventTypes::None;
    }
}

cudaDevice void IntConstraintsPropagator::updateDomains()
{
#ifdef GPU
    int vi = KernelUtils::getTaskIndex();
    if (vi >= 0 and vi < variables->count)
#else
    for (int vi = 0; vi < variables->count; vi += 1)
#endif
    {
        variables->domains.updateDomain(vi);
    }
}

cudaHostDevice void IntConstraintsPropagator::clearConstraintsToPropagate()
{
#if defined(GPU) && defined (__CUDA_ARCH__)
    int ci = KernelUtils::getTaskIndex();
    if (ci >= 0 and ci < constraints->count)
#else
    for (int ci = 0; ci < constraints->count; ci += 1)
#endif
    {
        constraintToPropagate[ci] = false;
    }
}

cudaDevice void IntConstraintsPropagator::checkEmptyDomains()
{
#ifdef GPU
    int vi = KernelUtils::getTaskIndex();
    if (vi >= 0 and vi < variables->count)
#else
    for (int vi = 0; vi < variables->count; vi += 1)
#endif
    {
        if (variables->domains.isEmpty(vi))
        {
            someEmptyDomain = true;
        }
    }
}

cudaDevice bool IntConstraintsPropagator::verifyConstraints()
{
    allConstraintsSatisfied = true;
#ifdef GPU
    Wrappers::checkSatisfiedConstraints<<<constraintsBlockCountDivergence, DEFAULT_BLOCK_SIZE>>>(this);
    hipDeviceSynchronize();
#else
    checkSatisfiedConstraints();
#endif

    return allConstraintsSatisfied;
}

cudaDevice void IntConstraintsPropagator::checkSatisfiedConstraints()
{
#ifdef GPU
    int ci = KernelUtils::getTaskIndex(true);
    if (ci >= 0 and ci < constraints->count)
#else
    for (int ci = 0; ci < constraints->count; ci += 1)
#endif
    {
        if (not constraints->satisfied(ci, variables))
        {
            allConstraintsSatisfied = false;
        }
    }
}
