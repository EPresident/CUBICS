#include <propagators/IntConstraintsPropagator.h>
#include <utils/Utils.h>
#include <wrappers/Wrappers.h>

void IntConstraintsPropagator::initialize(IntVariables* variables, IntConstraints* constraints)
{
    this->variables = variables;
    this->constraints = constraints;

    constraintToPropagate.initialize(constraints->count);
    constraintToPropagate.resize(constraints->count);
    clearConstraintsToPropagate();

#ifdef GPU
    constraintsBlockCountDivergence = KernelUtils::getBlockCount(constraints->count, DEFAULT_BLOCK_SIZE, true);
    constraintsBlockCount = KernelUtils::getBlockCount(constraints->count, DEFAULT_BLOCK_SIZE);
    variablesBlockCount = KernelUtils::getBlockCount(variables->count, DEFAULT_BLOCK_SIZE);
#endif
}

void IntConstraintsPropagator::deinitialize()
{
    constraintToPropagate.deinitialize();
}

cudaDevice bool IntConstraintsPropagator::propagateConstraints()
{
    someEmptyDomain = false;
    someConstraintsToPropagate = false;
#ifdef GPU
    Wrappers::setConstraintsToPropagate<<<constraintsBlockCount, DEFAULT_BLOCK_SIZE>>>(this);
    hipDeviceSynchronize();
#else
    setConstraintsToPropagate();
#endif

    while (someConstraintsToPropagate and (not someEmptyDomain))
    {
#ifdef GPU
        Wrappers::collectActions<<<constraintsBlockCountDivergence, DEFAULT_BLOCK_SIZE>>>(this);
        hipDeviceSynchronize();
#else
        collectActions();
#endif

#ifdef GPU
        Wrappers::clearDomainsEvents<<<variablesBlockCount, DEFAULT_BLOCK_SIZE>>>(this);
        hipDeviceSynchronize();
#else
        clearDomainsEvents();
#endif

#ifdef GPU
        Wrappers::updateDomains<<<variablesBlockCount, DEFAULT_BLOCK_SIZE>>>(this);
        hipDeviceSynchronize();
#else
        updateDomains();
#endif

#ifdef GPU
        Wrappers::clearConstraintsToPropagate<<<constraintsBlockCount, DEFAULT_BLOCK_SIZE>>>(this);
        hipDeviceSynchronize();
#else
        clearConstraintsToPropagate();
#endif

        someEmptyDomain = false;
#ifdef GPU
        Wrappers::checkEmptyDomains<<<variablesBlockCount, DEFAULT_BLOCK_SIZE>>>(this);
        hipDeviceSynchronize();
#else
        checkEmptyDomains();
#endif

        if (not someEmptyDomain)
        {
            someConstraintsToPropagate = false;
#ifdef GPU
            Wrappers::setConstraintsToPropagate<<<constraintsBlockCount, DEFAULT_BLOCK_SIZE>>>(this);
            hipDeviceSynchronize();
#else
            setConstraintsToPropagate();
#endif
        }
    }

    return (not someEmptyDomain);
}

/**
* Check if any constraint needs to be propagated, updating the appropriate
* flags.
* Propagation is required if any variable of a constraint has a "Changed" domain
* event.
* \see IntDomains
*/
cudaDevice void IntConstraintsPropagator::setConstraintsToPropagate()
{
#ifdef GPU
    int ci = KernelUtils::getTaskIndex();
    if (ci >= 0 and ci < constraints->count)
#else
    for (int ci = 0; ci < constraints->count; ci += 1)
#endif
    {
        for (int vi = 0; vi < constraints->variables[ci].size; vi += 1)
        {
            int event = variables->domains.events[constraints->variables[ci][vi]];

            if (event == IntDomains::EventTypes::Changed)
            {
                constraintToPropagate[ci] = true;
                someConstraintsToPropagate = true;
            }
        }
    }
}

/**
* Propagates all constraints flagged in "constraintToPropagate", and flips
* the respective flag.
*/
cudaDevice void IntConstraintsPropagator::collectActions()
{
#ifdef GPU
    int ci = KernelUtils::getTaskIndex(true);
    if (ci >= 0 and ci < constraints->count)
#else
    for (int ci = 0; ci < constraints->count; ci += 1)
#endif
    {
        if (constraintToPropagate[ci])
        {
            constraints->propagate(ci, variables);
            constraintToPropagate[ci] = false;
        }
    }
}

/** 
* Clears the domain events list.
* \see IntDomains
*/
cudaDevice void IntConstraintsPropagator::clearDomainsEvents()
{
#ifdef GPU
    int vi = KernelUtils::getTaskIndex();
    if (vi >= 0 and vi < variables->count)
#else
    for (int vi = 0; vi < variables->count; vi += 1)
#endif
    {
        variables->domains.events[vi] = IntDomains::EventTypes::None;
    }
}

/**
* Perform the domain reduction actions on all variables.
* \see IntDomains
*/
cudaDevice void IntConstraintsPropagator::updateDomains()
{
#ifdef GPU
    int vi = KernelUtils::getTaskIndex();
    if (vi >= 0 and vi < variables->count)
#else
    for (int vi = 0; vi < variables->count; vi += 1)
#endif
    {
        variables->domains.updateDomain(vi);
    }
}

/// Clears the "constraintToPropagate" vector.
cudaHostDevice void IntConstraintsPropagator::clearConstraintsToPropagate()
{
#if defined(GPU) && defined (__CUDA_ARCH__)
    int ci = KernelUtils::getTaskIndex();
    if (ci >= 0 and ci < constraints->count)
#else
    for (int ci = 0; ci < constraints->count; ci += 1)
#endif
    {
        constraintToPropagate[ci] = false;
    }
}

/// Checks if any domain has become empty, updating the "someEmptyDomain" flag.
cudaDevice void IntConstraintsPropagator::checkEmptyDomains()
{
#ifdef GPU
    int vi = KernelUtils::getTaskIndex();
    if (vi >= 0 and vi < variables->count)
#else
    for (int vi = 0; vi < variables->count; vi += 1)
#endif
    {
        if (variables->domains.isEmpty(vi))
        {
            someEmptyDomain = true;
        }
    }
}

/// \return true if all constraints are satisfied.
cudaDevice bool IntConstraintsPropagator::verifyConstraints()
{
    allConstraintsSatisfied = true;
#ifdef GPU
    Wrappers::checkSatisfiedConstraints<<<constraintsBlockCountDivergence, DEFAULT_BLOCK_SIZE>>>(this);
    hipDeviceSynchronize();
#else
    checkSatisfiedConstraints();
#endif

    return allConstraintsSatisfied;
}

/// Updates the "allConstraintsSatisfied" flag, scanning all constraints.
cudaDevice void IntConstraintsPropagator::checkSatisfiedConstraints()
{
#ifdef GPU
    int ci = KernelUtils::getTaskIndex(true);
    if (ci >= 0 and ci < constraints->count)
#else
    for (int ci = 0; ci < constraints->count; ci += 1)
#endif
    {
        if (not constraints->satisfied(ci, variables))
        {
            allConstraintsSatisfied = false;
        }
    }
}
