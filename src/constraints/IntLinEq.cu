#include "hip/hip_runtime.h"
#include <cmath>

#include <constraints/IntLinEq.h>
#include <data_structures/Vector.h>
#include <constraints/IntConstraints.h>

cudaDevice void IntLinEq::propagate(IntConstraints* constraints, int index, IntVariables* variables)
{
    Vector<int>* constraintVariables = &constraints->variables[index];
    Vector<int>* constraintParameters = &constraints->parameters[index];

    int sumPosCoeffLowValue = 0;
    int sumPosCoeffHighValue = 0;
    int sumNegCoeffLowValue = 0;
    int sumNegCoeffHighValue = 0;
    for (int i = 0; i < constraintVariables->size; i += 1)
    {
        int variableIndex = constraintVariables->at(i);
        int variableCoefficient = constraintParameters->at(i);

        if (variableCoefficient > 0)
        {
            sumPosCoeffLowValue += variableCoefficient * variables->domains.getMin(variableIndex);
            sumPosCoeffHighValue += variableCoefficient * variables->domains.getMax(variableIndex);
        }
        else
        {
            sumNegCoeffLowValue += (-variableCoefficient) * variables->domains.getMin(variableIndex);
            sumNegCoeffHighValue += (-variableCoefficient) * variables->domains.getMax(variableIndex);
        }
    }

    int b = constraintParameters->back();
    for (int i = 0; i < constraintVariables->size; i += 1)
    {
        int variableIndex = constraintVariables->at(i);
        int variableCoefficient = constraintParameters->at(i);

        if (variables->domains.isSingleton(variableIndex))
        {
            continue;
        }

        if (variableCoefficient > 0)
        {
            int variableLowContribution = variableCoefficient * variables->domains.getMin(variableIndex);
            int variableHightContribution = variableCoefficient * variables->domains.getMax(variableIndex);

            float alpha = (b - (sumPosCoeffLowValue - variableLowContribution) + sumNegCoeffHighValue) / static_cast<float>(variableCoefficient);
            float gamma = (b - (sumPosCoeffHighValue - variableHightContribution) + sumNegCoeffLowValue) / static_cast<float>(variableCoefficient);

            variables->domains.actions.removeAnyGreaterThan(variableIndex, static_cast<int>(floor(alpha)));
            variables->domains.actions.removeAnyLesserThan(variableIndex, static_cast<int>(ceil(gamma)));

        }
        else
        {
            int variableLowContribution = (-variableCoefficient) * variables->domains.getMin(variableIndex);
            int variableHightContribution = (-variableCoefficient) * variables->domains.getMax(variableIndex);

            float beta = (-b + sumPosCoeffLowValue - (sumNegCoeffHighValue - variableHightContribution)) / static_cast<float>(-variableCoefficient);
            float delta = (-b + sumPosCoeffHighValue - (sumNegCoeffLowValue - variableLowContribution)) / static_cast<float>(-variableCoefficient);

            variables->domains.actions.removeAnyLesserThan(variableIndex, static_cast<int>(ceil(beta)));
            variables->domains.actions.removeAnyGreaterThan(variableIndex, static_cast<int>(floor(delta)));
        }
    }
}

cudaDevice bool IntLinEq::satisfied(IntConstraints* constraints, int index, IntVariables* variables)
{
    //Satisfaction check is performed only when all variables is ground

    Vector<int>* constraintVariables = &constraints->variables[index];
    Vector<int>* constraintParameters = &constraints->parameters[index];

    for (int i = 0; i < constraintVariables->size; i += 1)
    {
        if (not variables->domains.isSingleton(constraintVariables->at(i)))
        {
            return true;
        }
    }

    int sum = 0;
    for (int i = 0; i < constraintVariables->size; i += 1)
    {
        sum += constraintParameters->at(i) * variables->domains.getMin(constraintVariables->at(i));
    }

    return sum == constraintParameters->back();
}
